#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

typedef struct {
    size_t n;
    int k, theta;
    float pi;
}foo_t;

__attribute__((noinline)) __device__ int add(int a, int b, int i, const foo_t &foo )
{
    //random ops to avoid inlining
    if(i<=0)
        return 0;
    int val = foo.n * foo.k;
    val += int(foo.pi);
    val -= foo.theta;
    val -= (a*b);
    return val;
}

__global__ void vecAdd(int *a, int *b, int *c, const foo_t foo)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (id < foo.n)
    {
        c[id] += add(a[id], b[id], id+1, foo);
    }
}

int main( int argc, char* argv[] )
{
    // Size of vectors
    const int n = 10000;

    // Host input vectors
    int *h_a;
    int *h_b;
    //Host output vector
    int *h_c;

    // Device input vectors
    int *d_a;
    int *d_b;
    //Device output vector
    int *d_c;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(int);

    foo_t foo = {.n=bytes, .k = n, .theta = 90, .pi = 3.14};

    // Allocate memory for each vector on host
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }

    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 512;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, foo);

    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );

    // Sum up vector c and print result divided by n, this should equal 1 within error
    int sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    //printf("final result: %f\n", sum/n);

    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
